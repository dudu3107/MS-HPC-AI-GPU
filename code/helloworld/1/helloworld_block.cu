/*
 * How to build:
 *
 * nvcc -arch=sm_80 -o helloworld_block helloworld_block.cu
 *
 * Note that you need to adjust the architecture version to your current GPU hardware.
 * Hardware version can be probed with e.g. deviceQuery example (from Nvidia SDK samples).
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void hello()
{
  printf("I'm a thread %d in block %d\n",
         threadIdx.x, blockIdx.x);
}


int main(int argc,char **argv)
{

  // default values for 
  // - gridSize :  number of blocks
  // - blockSize : number of threads per block

  // unsigned int gridSize  = argc > 1 ? atoi(argv[1]) : 1;
  // unsigned int blockSize = argc > 2 ? atoi(argv[2]) : 16;

  dim3 blockSize (4, 4);
  dim3 gridSize (1);

  // launch the kernel
  hello<<<gridSize, blockSize>>>();
  
  // force the printf()s to flush
  hipDeviceSynchronize();
  
  printf("That's all!\n");
  
  return EXIT_SUCCESS;
}
