// nvcc helloworld.cu -o helloworld


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu(void) {
  printf("Hello World from GPU thread %d, block %d !\n",
         threadIdx.x, blockIdx.x);
}

int main(int argc, char* argv[]) {
  printf("Hello from CPU !\n");
  print_from_gpu<<<1,1>>>();
  //cudaDeviceSynchronize();
  return EXIT_SUCCESS;
}
