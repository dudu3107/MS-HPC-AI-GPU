#include "hip/hip_runtime.h"
/**
 * \file heat2d_solver_gpu_naive.cu
 * \brief Solve 2D heat equation (finite difference method). GPU version (naive).
 *
 * We solve the 2D Heat equation \f$\partial_t \phi = \alpha \left[
 * \partial^2_x \phi + \partial^2_y \phi \right] \f$, \f$ 0 \leq x
 * \leq L_x \f$, \f$ 0 \leq y \leq L_y \f$, \f$ 0 \leq t\f$.\\
 *
 * Method : Finite Difference, FTCS scheme
 *
 * GPU Features: use only global memory
 *
 * boundary condition : Dirichlet
 *
 * GPU version : naive
 *
 * \date 17-dec-2009.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sys/time.h> // for gettimeofday

#ifdef _OPENMP
#include <omp.h>
#endif
#include "openmp_utils.h"

// includes, project
//#include <helper_functions.h>
#include "cuda_error.h"
#include "CudaTimer.h"
#include "Timer.h"

// parameters and real_t typedef
#include "param.h"

// for output results
#include "output.h"

// GPU solver
#include "heat2d_kernel_gpu_naive.cu"

// CPU solver
#include "heat_kernel_cpu.h"

// initial conditions
#include "misc.h"

// cuda helper
#include "cuda_helper.h"

/////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest( int argc, char** argv);

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv) 
{
  int status = runTest(argc, argv);

  CUDA_API_CHECK ( hipDeviceSynchronize() );
  CUDA_API_CHECK ( hipDeviceReset() );

  return status;
}

/////////////////////////////////////////////////////////////////////////
//! Run solver on GPU
/////////////////////////////////////////////////////////////////////////
int
runTest(int argc, char** argv) 
{
  int devID;
  hipDeviceProp_t deviceProps;
  
  devID = findCudaDevice(argc, (const char **)argv);
  
  // get number of SMs on this GPU
  CUDA_API_CHECK( hipGetDeviceProperties(&deviceProps, devID) );
  printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

  /*
   * read and print parameters
   */
  // default parameter file
  std::string paramFile("heatEqSolver.par");

  // if argv[1] exists use it as a parameter file
  if (argc>1) {
    printf("trying to read parameters from file %s ...\n",argv[1]);
    paramFile = std::string(argv[1]);
  }

  // read parameter file
  readParamFile(paramFile);

  // print parameters on screen
  printParameters("HEAT 2D - GPU (NAIVE)");

  CudaTimer gpuTimer;

  unsigned int mem_size = sizeof(real_t)*NX*NY;

  // allocate host memory
  real_t* data1 = (real_t*) malloc( mem_size);
  real_t* data2 = (real_t*) malloc( mem_size);
  
  ///////////////////////////////////////////////////
  // compute GPU solution to 2D heat equation
  ///////////////////////////////////////////////////
  
  // inital condition
  initCondition2D (data1);

  // allocate device memory
  real_t* d_data1;
  real_t* d_data2;

  // device memory allocation (using hipMalloc)
  CUDA_API_CHECK( hipMalloc( (void**)&d_data1, N*sizeof(real_t) ) );
  CUDA_API_CHECK( hipMalloc( (void**)&d_data2, N*sizeof(real_t) ) );

  // copy host memory to device
  CUDA_API_CHECK( hipMemcpy( d_data1, data_1, N*sizeof(real_t), hipMemcpyHostToDevice) );
  //CUDA_API_CHECK( hipMemcpy( d_data2, data_2, N*sizeof(real_t), hipMemcpyHostToDevice) );
    
   
  // setup execution parameters for cuda kernel
  // grid dimension for naive kernel
  unsigned int threadsPerBlockX=16;
  unsigned int threadsPerBlockY=16;
  dim3  threads(4, 4);
  dim3  grid(/* TODO */);
    
  printf("grid  size : %u %u\n",grid.x,grid.y);
  printf("block size : %u %u\n",threads.x,threads.y);

  // start timer
  gpuTimer.start();

  // time loop executing naive kernel
  int iTime   =  0;
  int iOutput = -1;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {

    real_t* data_in  = iTime%2 == 0 ? d_data1 : d_data2;
    real_t* data_out = iTime%2 == 0 ? d_data2 : d_data1;

    if (useOrder2) { // use the 2nd order accurate scheme
      
      heat2d_ftcs_naive_order2_kernel<<< grid, threads >>>( data_in, 
                                                            data_out,
							    NX, NY,
							    o2.R, o2.R2);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    } else if (useOrder2b) { // use the 2nd order accurate scheme
      
      heat2d_ftcs_naive_order2b_kernel<<< grid, threads >>>( data_in, 
                                                             data_out,
							     NX, NY,
							     o2.R, o2.R2b);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    } else { // use the 4th order accurate scheme
      
      heat2d_ftcs_naive_order4_kernel<<< grid, threads >>>( data_in,
                                                            data_out,
							    NX, NY,
							    o4.S, o4.S2);
      CUDA_KERNEL_CHECK("Kernel execution failed");
      
    }

    /* save output (just for cross-checking, do not save when
       measuring computing time */
    if (ENABLE_GPU_SAVE) {

      if (iTime%T_OUTPUT == 0) {
	iOutput++;
	CUDA_API_CHECK( hipMemcpy( /* TODO */ ) );      
      }
      // PGM output
      if (SAVE_PGM and iTime%T_OUTPUT == 0)
	save_pgm(data1, "heat2d_gpu_naive_",iOutput,NX,NY);
      
      // MathGL save (3D view)
      if (SAVE_MGL and iTime%T_OUTPUT == 0)
	save_mgl(data1, "heat2d_gpu_naive_",iOutput,NX,NY);

      // VTK output
      if (SAVE_VTK and iTime%T_OUTPUT == 0)
	save_vtk(data1, "heat2d_gpu_naive_",iOutput);

      // HDF5 output
      if (SAVE_HDF5 and iTime%T_OUTPUT == 0)
	save_hdf5(data1, "heat2d_gpu_naive_",iOutput);

    }

  } // end for loop
  
  // stop timer
  gpuTimer.stop();

  real_t gpu_time = gpuTimer.elapsed();
  printf( "GPU Processing time: %f (s)\n", gpu_time);
  
  // copy result from device to host
  real_t *resGPU = (real_t*) malloc( mem_size);
  CUDA_API_CHECK( hipMemcpy( /* TODO */ ) );
    
  if (SAVE_HDF5)
    write_xdmf_wrapper("heat2d_gpu_naive",N_ITER,T_OUTPUT);
 
  ////////////////////////////////////////////////////////
  // compute reference (CPU) solution to 2D heat equation
  // for performance comparison
  ////////////////////////////////////////////////////////
  printf("compute CPU reference solution\n");
  initCondition2D (data1);
  initCondition2D (data2);

  print_openmp_status();

  Timer cpuTimer;
  cpuTimer.start();
  
  // time loop
  iTime=0;
  for (iTime=0; iTime*DT<TMAX; ++iTime) {
    
    real_t* data_in  = iTime%2 == 0 ? data1 : data2;
    real_t* data_out = iTime%2 == 0 ? data2 : data1;

    if (useOrder2) {
    
      heat2d_ftcs_cpu_order2( data_in, data_out );
      
    } else if (useOrder2b) {

      heat2d_ftcs_cpu_order2b( data_in, data_out );
      
    } else {

      heat2d_ftcs_cpu_order4( data_in, data_out );
      
    }
  }

  // stop timer
  cpuTimer.stop();
  real_t cpu_time = cpuTimer.elapsed();
  
  printf( "CPU Processing time: %g (s)\n", cpu_time);
  printf( "Speedup GPU/CPU : %f\n",cpu_time/gpu_time);

  printf("...comparing the results\n");
  double sum = 0, delta = 0;
  for(unsigned i = 0; i < NX*NY; i++){
    delta += (resGPU[i] - data1[i]) * (resGPU[i] - data1[i]);
    sum   += data1[i] * data1[i];
  }
  double L2norm = sqrt(delta / sum);
  printf("iteration %d relative L2 norm: %E\n", iTime, L2norm);

  // cuda device prop (to compute max bandwidth
  hipDeviceProp_t deviceProp;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&deviceProp, deviceId);
  //printf("GPU mem clock rate in kHz %d\n",deviceProp.memoryClockRate);
  //printf("GPU mem bus width %d\n",deviceProp.memoryBusWidth);
  real_t gpuMaxBW = 1e-9*deviceProp.memoryClockRate*1000*(deviceProp.memoryBusWidth/8)*2;
  
  // bandwidth
  double totalBytes;
  if (useOrder2)
    totalBytes = NX*NY*sizeof(real_t) * (5 + 1) * iTime;
  else if (useOrder2b)
    totalBytes = NX*NY*sizeof(real_t) * (3*3 + 1) * iTime;
  else
    totalBytes = NX*NY*sizeof(real_t) * (9 + 1) * iTime;
  printf("CPU Bandwidth %f GBytes/s\n", totalBytes/cpu_time*1e-9);
  printf("GPU Bandwidth %f GBytes/s out of %f (%6.2f %%)\n", totalBytes/gpu_time*1e-9,gpuMaxBW,totalBytes/gpu_time*1e-9/gpuMaxBW*100);

  // cleanup memory
  free(data1);
  free(data2);
  free(resGPU);
  
  CUDA_API_CHECK( hipFree(d_data1) );
  CUDA_API_CHECK( hipFree(d_data2) );
  
  return EXIT_SUCCESS;
}
