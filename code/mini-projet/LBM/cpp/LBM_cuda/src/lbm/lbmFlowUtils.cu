#include "hip/hip_runtime.h"
#include <math.h> // for M_PI = 3.1415....

#include "lbmFlowUtils.h"

#include "lbmFlowUtils_kernels.h"
#include "cuda_error.h"

// ======================================================
// ======================================================
void macroscopic(const LBMParams& params, 
                 const velocity_array_t v,
                 const real_t* fin_d,
                 real_t* rho_d,
                 real_t* ux_d,
                 real_t* uy_d)
{

  const int nx = params.nx;
  const int ny = params.ny;
  //const int npop = LBMParams::npop;

  // TODO : call kernel
  unsigned int threadsPerBlockX=32;
  unsigned int threadsPerBlockY=8;

  dim3  threads(threadsPerBlockX, threadsPerBlockY, 1);
  dim3  gridSize( (nx+threads.x-1)/threads.x, (ny+threads.y-1)/threads.y , 1);

  macroscopic_kernel<<<gridSize,threads>>>(params, v, fin_d, rho_d, ux_d, uy_d);
  CUDA_KERNEL_CHECK("macroscopic_kernel");


} // macroscopic

// ======================================================
// ======================================================
void equilibrium(const LBMParams& params, 
                 const velocity_array_t v,
                 const weights_t t,
                 const real_t* rho_d,
                 const real_t* ux_d,
                 const real_t* uy_d,
                 real_t* feq_d)
{

  const int nx = params.nx;
  const int ny = params.ny;
  const int npop = LBMParams::npop;

  // TODO : call kernel
  unsigned int threadsPerBlockX=32;
  unsigned int threadsPerBlockY=8;
  //unsigned int threadsPerBlockZ=32;

  dim3  threads(threadsPerBlockX, threadsPerBlockY, 1);//threadsPerBlockZ);
  dim3  gridSize( (nx+threads.x-1)/threads.x, (ny+threads.y-1)/threads.y , 1);//(npop+threads.z-1)/threads.z);

  equilibrium_kernel<<<gridSize,threads>>>(params, v, t, rho_d, ux_d, uy_d, feq_d);
  CUDA_KERNEL_CHECK("equilibrium_kernel");

} // equilibrium

// ======================================================
// ======================================================
void init_obstacle_mask(const LBMParams& params, 
                        int* obstacle, 
                        int* obstacle_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  const real_t cx = params.cx;
  const real_t cy = params.cy;

  const real_t r = params.r;

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      real_t x = 1.0*i;
      real_t y = 1.0*j;

      obstacle[index] = (x-cx)*(x-cx) + (y-cy)*(y-cy) < r*r ? 1 : 0;

    } // end for i
  } // end for j

  // TODO : copy host to device
  CUDA_API_CHECK( hipMemcpy( obstacle_d, obstacle, nx*ny*sizeof(int), hipMemcpyHostToDevice ) );

} // init_obstacle_mask

// ======================================================
// ======================================================
__host__ __device__
real_t compute_vel(int dir, int i, int j, real_t uLB, real_t ly)
{

  // flow is along X axis
  // X component is non-zero
  // Y component is always zero

  return (1-dir) * uLB * (1 + 1e-4 * sin(j/ly*2*M_PI));

} // compute_vel

// ======================================================
// ======================================================
void initialize_macroscopic_variables(const LBMParams& params, 
                                      real_t* rho, real_t* rho_d,
                                      real_t* ux, real_t* ux_d,
                                      real_t* uy, real_t* uy_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      rho[index] = 1.0;
      ux[index]  = compute_vel(0, i, j, params.uLB, params.ly);
      uy[index]  = compute_vel(1, i, j, params.uLB, params.ly);

    } // end for i
  } // end for j

  // TODO : copy host to device
  CUDA_API_CHECK( hipMemcpy( rho_d, rho, nx*ny*sizeof(real_t), hipMemcpyHostToDevice ) );
  CUDA_API_CHECK( hipMemcpy( ux_d, ux, nx*ny*sizeof(real_t), hipMemcpyHostToDevice ) );
  CUDA_API_CHECK( hipMemcpy( uy_d, uy, nx*ny*sizeof(real_t), hipMemcpyHostToDevice ) );


} // initialize_macroscopic_variables

// ======================================================
// ======================================================
void border_outflow(const LBMParams& params, real_t* fin_d)
{

  // TODO : call kernel
  const int ny = params.ny;
  unsigned int threadsPerBlock=256;
  dim3  threads(1, threadsPerBlock, 1);
  dim3  gridSize( 1, (ny+threads.y-1)/threads.y , 1);

  border_outflow_kernel<<<gridSize,threads>>>(params, fin_d);
  CUDA_KERNEL_CHECK("border_outflow_kernel");

} // border_outflow


// ======================================================
// ======================================================
void border_inflow(const LBMParams& params, const real_t* fin_d, 
                   real_t* rho_d, real_t* ux_d, real_t* uy_d)
{
  // TODO : call kernel
  const int ny = params.ny;
  unsigned int threadsPerBlock=256;
  dim3  threads(1, threadsPerBlock, 1);
  dim3  gridSize( 1, (ny+threads.y-1)/threads.y , 1);

  border_inflow_kernel<<<gridSize,threads>>>(params, fin_d, rho_d, ux_d, uy_d);
  CUDA_KERNEL_CHECK("border_inflow_kernel");

} // border_inflow


// ======================================================
// ======================================================
void update_fin_inflow(const LBMParams& params, const real_t* feq_d, 
                       real_t* fin_d)
{

  // TODO : call kernel
  const int ny = params.ny;
  unsigned int threadsPerBlock=256;
  dim3  threads(1, threadsPerBlock, 1);
  dim3  gridSize( 1, (ny+threads.y-1)/threads.y , 1);

  update_fin_inflow_kernel<<<gridSize,threads>>>(params, feq_d, fin_d);
  CUDA_KERNEL_CHECK("update_fin_inflow_kernel");

} // update_fin_inflow


// ======================================================
// ======================================================
void compute_collision(const LBMParams& params, 
                       const real_t* fin_d,
                       const real_t* feq_d,
                       real_t* fout_d)
{

  const int nx = params.nx;
  const int ny = params.ny;
  const int npop = LBMParams::npop;

  // TODO : call kernel
  unsigned int threadsPerBlockX=32;
  unsigned int threadsPerBlockY=8;
  //unsigned int threadsPerBlockZ=3;

  dim3  threads(threadsPerBlockX, threadsPerBlockY, 1);//threadsPerBlockZ);
  dim3  gridSize( (nx+threads.x-1)/threads.x, (ny+threads.y-1)/threads.y , 1);//(npop+threads.z-1)/threads.z);

  compute_collision_kernel<<<gridSize,threads>>>(params, fin_d, feq_d, fout_d);
  CUDA_KERNEL_CHECK("compute_collision_kernel");

} // compute_collision

// ======================================================
// ======================================================
void update_obstacle(const LBMParams &params, 
                     const real_t* fin_d,
                     const int* obstacle_d, 
                     real_t* fout_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  // TODO : call kernel
  unsigned int threadsPerBlockX=32;
  unsigned int threadsPerBlockY=8;

  dim3  threads(threadsPerBlockX, threadsPerBlockY, 1);
  dim3  gridSize( (nx+threads.x-1)/threads.x, (ny+threads.y-1)/threads.y , 1);

  update_obstacle_kernel<<<gridSize,threads>>>(params, fin_d, obstacle_d, fout_d);
  CUDA_KERNEL_CHECK("update_obstacle_kernel");

} // update_obstacle

// ======================================================
// ======================================================
void streaming(const LBMParams& params,
               const velocity_array_t v,
               const real_t* fout_d,
               real_t* fin_d)
{

  const int nx = params.nx;
  const int ny = params.ny;
  const int npop = LBMParams::npop;

  // TODO : call kernel
  unsigned int threadsPerBlockX=32;
  unsigned int threadsPerBlockY=8;
  //unsigned int threadsPerBlockZ=3;

  dim3  threads(threadsPerBlockX, threadsPerBlockY, 1);//threadsPerBlockZ);
  dim3  gridSize( (nx+threads.x-1)/threads.x, (ny+threads.y-1)/threads.y , 1);//(npop+threads.z-1)/threads.z);

  streaming_kernel<<<gridSize,threads>>>(params, v, fout_d, fin_d);
  CUDA_KERNEL_CHECK("streaming_kernel");

} // streaming
